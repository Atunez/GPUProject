#include "hip/hip_runtime.h"
/*
  Brady Adcock
  Abdel Issa 

  Implementing cool paper on range query
*/

#include <math.h>
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "CHECK.h"
#include "d_rangequery.h"
#include "config.h"  //defines THREADSPERBLOCK
#include "wrappers.h"

// prototype for kernels
static __global__ void decomp(ulong*, ulong, ulong);
static __global__ void decompress(ulong*, ulong*, ulong*, ulong, int);

/*
 Given an array of relevant cols (representing bins which
 classify the variable of interest) return resultant 
 bit vector R. 

 params,
 	cols:   array of bitvectors representing bins of variable 
 		being queried. 2d array reprsented as 1d array because
		each element of the array is variable length.  
	cSizes: array desciribing the size of each compressed col
		in terms of number of 64 bit words.  
	dSizes: number of 64 bit words in the decompressed cols
 return,
	R:	bitvector representing rows who match range query.
*/
void d_decompress (ulong * cols, ulong * cSizes, ulong * dData, ulong dSize, int numCols)
{
	ulong * R; 
	ulong * d_cols; // 1d array representing the 2d array of compressed bins
	ulong * d_cSizes;  
	ulong * d_dData; // 1d array representing the 2d array of decompressed bins
	ulong totalSize =  0;
	int i;

	// determine total size of cols (cData)
	for(i = 0; i < numCols; i++) 
	{
		totalSize += cSizes[i];
	}
	totalSize *= sizeof(ulong); 

	// malloc cData && dData
	CHECK(hipMalloc((void **) &d_cols, totalSize));
	CHECK(hipMalloc((void **) &d_dData, dSize * sizeof(ulong)));
	// malloc cSizes
	CHECK(hipMalloc((void **) &d_cSizes, numCols * sizeof(ulong)));
	// malloc R (same as dSize)
	CHECK(hipMalloc((void **) &R, dSize * sizeof(ulong)));

	// copy cData over to device
	CHECK(hipMemcpy(d_cols, cols, totalSize, hipMemcpyHostToDevice));
	// copy cSizes over to device
	CHECK(hipMemcpy(d_cSizes, cSizes, numCols * sizeof(ulong), hipMemcpyHostToDevice));

	// make grid && block 
	dim3 grid(1, 1, 1); 		// only need a few threads because this 
	dim3 block(numCols, 1, 1);	// kernel will launch many more

	// launch kernel decompress
	decompress<<<grid, block>>>(d_cols, d_cSizes, d_dData, dSize, numCols);

	// copy decompressed data back from device
	CHECK(hipMemcpy(dData, d_dData, dSize * sizeof(ulong), hipMemcpyDeviceToHost));

	// hipFree everything on device
	CHECK(hipFree(d_cols));
	CHECK(hipFree(d_cSizes));
	CHECK(hipFree(d_dData));
}

/*
 Given pointer to array of WAH compressed bitvectors, decompress them
 by launching decomp kernels.

 params,
	cols:	array of bitvectors with WAH 64 bit encoding
*/
__global__ void decompress (ulong * cols, ulong * cSizes, ulong * dData, ulong dSize, int numCols) 
{
	int col = threadIdx.x;
	ulong col_offset = 0;		
	ulong * bitVec;
	int i;

	// compute pointer arithmetic
	for(i = 0; i < col; i++)
	{
		col_offset += cSizes[i]; 
	}
	// bitVec = pointer to cData to be processed by decomp kernel
	bitVec = cols + col_offset;

	// create grid and block according to cSize
	dim3 grid(ceil(1.0 * cSizes[col]/THREADSPERBLOCK), 1, 1);
	dim3 block(THREADSPERBLOCK, 1, 1);

	// launch decomp kernel
	decomp<<<grid, block>>>(bitVec, cSizes[col], dSize); 	
}

/*
 Given a bitvector representing a single bin, decompress its
 data and return it.

 params,
	cData:	compressed WAH vector
	cSize:	number of 64 bit words CData represents
	dSize:	number of 64 bit words in orignal (decompressed) data 
*/
__global__ void decomp(ulong * cData, ulong cSize, ulong dSize)
{
	// debugging...
	printf("cData[0]: %lu, cSize: %lu, dSize: %lu\n", cData[0], cSize, dSize);
	
	// create index in cData
	uint cWordIndex = blockIdx.x * THREADSPERBLOCK + threadIdx.x;

	// bounds checking
	if (cWordIndex >= cSize) { return; }

	
	
//-------------------------

	// TODO: for each 64-bit WAH word in cData check word type
		// update DecompSize[index] with 1:lit or len:fill 
			// PARALLIZATION OPPORTUNITY

	// TODO: create startingPoints array using exclusive scan of DecompSize[]

	// TODO: create endPoints[dSize], init with 0's

	// TODO: for each 64-bit WAH word in cData add entry into endPoints
		// endPoints[startingPoints[i]-1] = 1
			// PARALLIZATION OPPORTUNITY

	// TODO: create wordIndex to store index of atom in cData that contains data
	// for the wordIndex[i] decompressed word
		// exclusive scan on endPoints
			// PARALLIZATION OPPORTUNITY

	// TODO: for each word in the decompressed data : decompress
		// grab tempWord, the cData word encoding this dData word
		// if tempWord is a literal then update dData[i] with tempWord
		// else fill dData[i] with all 0's or one 0 and sixtythree 1's
}
	
